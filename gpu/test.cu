#include "hip/hip_runtime.h"
#include "../common/headers/arguments.h"

//INCLUDE INSTANCES
#include "../bounds/headers/libbounds.h"

#include "../common/headers/pbab.h"
#include "../common/headers/solution.h"
#include "../common/headers/ttime.h"
#include "../common/headers/log.h"

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#include "./headers/gpuerrchk.h"
#include "./headers/gpu_helper.cuh"
#include "./headers/gpu_fsp_bound_weak.cuh"


void initializeFSP_LB(pbab* pbb)
{
    (pbb->instance->data)->seekg(0);
    (pbb->instance->data)->clear();

    *(pbb->instance->data) >> nbJob_h;
    *(pbb->instance->data) >> nbMachines_h;

    allocate_host_bound_tmp();

    for (int i = 0; i < nbMachines_h; i++) {
        for (int j = 0; j < nbJob_h; j++)
            *(pbb->instance->data) >> tempsJob_h[i * nbJob_h + j];
        fillMinTempsArrDep();
        fillSumPT();
    }

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_sumPT), sumPT_h, nbMachines_h * sizeof(int)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_minTempsDep), minTempsDep_h, nbMachines_h * sizeof(int)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_minTempsArr), minTempsArr_h, nbMachines_h * sizeof(int)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_tempsJob), tempsJob_h, nbJob_h * nbMachines_h * sizeof(int)));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_nbMachines), &nbMachines_h, sizeof(int)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_nbJob), &nbJob_h, sizeof(int)));

    // gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_sum), &somme_h, sizeof(int)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(size_d), &nbJob_h, sizeof(int)));
    // gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(_nbJobPairs), &nbJobPairs_h, sizeof(int)));

    free_host_bound_tmp();
}

int main(int argc,char *argv[])
{
    arguments::readIniFile();
    arguments::parse_arguments(argc, argv);
    arguments::initialize();

    pbab * pbb = new pbab();

    FILELog::ReportingLevel() = logINFO;
    FILE* log_fd = fopen( "./logs/test.txt", "w" );
    Output2FILE::Stream() = log_fd;

    //========================================
    //host lower bound
    bound_abstract * bound;
    if (arguments::problem[0] == 'f') {
        bound = new bound_fsp_weak();
        bound->set_instance(pbb->instance);
        bound->init(arguments::branchingMode,0,0);
    }
    //======================================

    //init device, allocate memory...
    gpuErrchk(hipSetDevice(0));
    gpuErrchk(hipFree(0));
    initializeFSP_LB(pbb);

    int *schedule;
    int *lim1;
    int *costs;
    int *state;
    int nbIVM = arguments::nbivms_gpu;

	gpuErrchk(hipMallocManaged(&schedule,nbIVM*nbJob_h*sizeof(int)));
	gpuErrchk(hipMallocManaged(&costs,nbIVM*2*nbJob_h*sizeof(int)));
	gpuErrchk(hipMallocManaged(&lim1,nbIVM*sizeof(int)));
	gpuErrchk(hipMallocManaged(&state,nbIVM*sizeof(int)));

    int nbChildren=0;
    //creating some subproblems (only begin-scheduling)
    for(int i=0;i<nbIVM;i++)
    {
        for(int j=0;j<nbJob_h;j++){
            schedule[i*nbJob_h+j]=j;
            costs[i*2*nbJob_h+j]=0;
        }
        helper::shuffle(schedule,nbJob_h);
        state[i]=1;
        lim1[i]=helper::intRand(-1,nbJob_h-5);//-1 means no job scheduled...

        nbChildren += (nbJob_h-lim1[i]-1);
    }

    struct timespec startt,endt;
    clock_gettime(CLOCK_MONOTONIC,&startt);

    //use 32 threads per subproblem to evaluate all children nodes
    int NN=4;
    size_t smem = (NN * (nbJob_h + 3 * nbMachines_h)) * sizeof(int);
    boundWeak_Begin << < (nbIVM+NN-1) / NN, NN * 32, smem >>>
    (schedule, lim1, costs, state);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    clock_gettime(CLOCK_MONOTONIC,&endt);
    auto t=(endt.tv_sec-startt.tv_sec)+(endt.tv_nsec-startt.tv_nsec)/1e9;
    std::cout<<"Computed " <<nbChildren << " Child subproblems in: "<<t<<" sec"<<std::endl;
    std::cout<<"Per LB: " <<(float)nbChildren/t<<std::endl;

    //===============================
    clock_gettime(CLOCK_MONOTONIC,&startt);

    boundWeak_Begin_One <<< (nbIVM+127) / 128, 128 >>> (schedule, lim1, costs, state);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    clock_gettime(CLOCK_MONOTONIC,&endt);
    t=(endt.tv_sec-startt.tv_sec)+(endt.tv_nsec-startt.tv_nsec)/1e9;
    std::cout<<"Computed " <<nbIVM << " Parent subproblems in: "<<t<<" sec"<<std::endl;
    std::cout<<"Per LB: " <<(float)nbIVM/t<<std::endl;

    int *costsCPU=new int[nbIVM];

    int c[2];
    for(int i=0;i<nbIVM;i++)
    {
        bound->bornes_calculer(&schedule[i*nbJob_h], lim1[i], nbJob_h,c, 999999);
        costsCPU[i]=c[0];
    }

    bool ok=true;
    for(int i=0;i<nbIVM;i++)
    {
        ok &= (costs[i]==costsCPU[i]);
    }
    if(!ok)
    {
        for(int i=0;i<nbIVM;i++)
            printf("%2d\t %2d %4d\t %4d\n",i,lim1[i],costs[i],costsCPU[i]);
    }else{
        printf("test passed");
    }
    printf("\n");


    free(costsCPU);
    hipFree(schedule);
    hipFree(costs);
    hipFree(lim1);
    hipFree(state);

}
